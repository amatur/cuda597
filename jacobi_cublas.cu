#include "hip/hip_runtime.h"
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <cstring>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/iterator/zip_iterator.h>
//using namespace std;

void init1d(float **A, int n){
	float* B = (float*) calloc(n,sizeof(float));
	*A = B;
}

float getError(float *x, float *xnew, int N)
{

  float *d_x;
  float *d_y;
    float *yy;
    init1d(&yy, N);
  cublasInit();
  hipblasSetVector(N, sizeof(x[0]), x, 1, d_x, 1);
  hipblasSetVector(N, sizeof(xnew[0]), xnew, 1, d_y, 1);

  hipblasSaxpy(N, -1.0f, d_x, 1, d_y, 1);

  hipblasGetVector(N, sizeof(yy[0]), d_y, 1, y, 1);
  float sum = hipblasSasum (N,  d_y, 1);
  cublasShutdown();
  return sum;
}

template<typename T>
class square_diff_thr : public thrust::unary_function<thrust::tuple<T, T>, T>
{
public:
    __host__ __device__
    T operator()(const thrust::tuple<T, T>& x) const {
        return (thrust::get<1>(x) - thrust::get<0>(x)) *
            (thrust::get<1>(x) - thrust::get<0>(x));
    }
};

template<typename T>
class square_thr : public thrust::unary_function<T, T>
{
public:
    __host__ __device__
    T operator()(const T& x) const {
        return x*x;
    }
};

template<typename T>
T getErrorThrust(T *A_d, T *B_d, int N)
{
    typedef thrust::device_ptr<T> FloatIterator;
    typedef thrust::tuple<FloatIterator, FloatIterator> IteratorTuple;
    typedef thrust::zip_iterator<IteratorTuple> ZipIterator;

    thrust::device_ptr<T> A_ptr(A_d);
    thrust::device_ptr<T> B_ptr(B_d);

    ZipIterator first =
        thrust::make_zip_iterator(thrust::make_tuple(A_ptr, B_ptr));
    ZipIterator last =
        thrust::make_zip_iterator(thrust::make_tuple(A_ptr + N*N,
                                                     B_ptr + N*N));

    T a1 = thrust::transform_reduce(first, last, square_diff_thr<T>(),
                                  static_cast<T>(0), thrust::plus<T>());
    T a2 = thrust::transform_reduce(B_ptr, B_ptr + N*N,
                                  square_thr<T>(), static_cast<T>(0),
                                  thrust::plus<T>());

    return sqrt(a1/a2);
}



void init2d(float ***A, int n){
	float** B = (float**) calloc(n,sizeof(float*));
    for(int i =0; i <n; i++){
		B[i] = (float*) calloc(n,sizeof(float));
	}
	*A = B;
}

void init2d(float ***A, float ***A2, int n){
	float** B = (float**) calloc(n,sizeof(float*));
    for(int i =0; i <n; i++){
		B[i] = (float*) calloc(n,sizeof(float));

	}
	for (int y = 0; y < n; y++) {
		for (int x = 0; x < n; x++) {
            B[y][x] = *A2[y][x];
		}
	}
	*A = B;
}





void print(float *mat, int numRows){

	for (int x = 0; x < numRows; x++) {
		printf("%-20.3f ", mat[x]);
	}
	printf("\n");

}

void print(float **mat, int numRows){
	for (int y = 0; y < numRows; y++) {
		for (int x = 0; x < numRows; x++) {
            printf("%-20.3f ", mat[y][x]);
		}
		printf("\n");
	}
	printf("\n");
}



void convertTo1D(float** A, float* A_1d, int N){
	int k    = 0;
	for(int irow=0; irow<N; irow++)
	for(int icol=0; icol<N; icol++)
	A_1d[k++] = A[irow][icol];
}


void jacobiSolve(int n, float** A, float* B, float* x, float eps = 1e-10, int maxit = 100){
	memset(x, 0, n*sizeof(*x)); //init guess

	//random initialization
	for(int j=0; j<n; j++)
	{
		x[j] = (float)rand()/(float)(RAND_MAX)*1.0;
	}

	float* sigma = (float*) calloc(n,sizeof(float));

	float* y = (float*) calloc(n,sizeof(float));

	//float *C = (float *) malloc( n * sizeof(float));
	int it = 0;

	int k = 0;
	do{
		it++;

		float totSum = 0.0;
		float localSum = 0.0;
		float localInd = 0.0;
		for (int i=0; i<n; i++) {
			sigma[i] = B[i];
			for (int j = 0; j < n; j++) {
				//if(j!=1){
				sigma[i] -= A[i][j] * x[j];

				//}
			}
			sigma[i] /= A[i][i];

			y[i] += sigma[i];



			// Create a residual from part of the domain (when the indices  are 0, 5, 10, ...)
			if ( (i % 5) == 0)
			{
				localSum += ( (sigma[i] >= 0.0) ? sigma[i] : -sigma[i]);

			}

			// Create a residual from a single point
			if (i == n/2)
			{
				localInd += ( (sigma[i] >= 0.0) ? sigma[i] : -sigma[i]);
			}

			// Create a residual over all of the domain
			totSum += ( (sigma[i] >= 0.0) ? sigma[i] : -sigma[i]);
		}
		k = k + 1;
		//print(x, n);
		//printf("%f", getError(A, B, C, x, n));



		// Update x
		for(int i=0; i<n; i++) x[i] = y[i];

		// Print the residuals to the screen
		//printf("%4d\t%.3e\t%.3e\t%.3e \n",k,totSum,localSum,localInd);
		//getError(A, B, C, x, n)
		if(totSum <=eps || it >= maxit){
			break;
		}


	}while(true);

	free(sigma);
	free(y);
}



/**
* @brief   Randomly Initialize the A matrix
*/
void fillA_random(float **A, int n){
	int countA,countB;
	for (countA=0; countA<n; countA++)
	{
		for (countB=0; countB<n; countB++)
		{
			A[countA][countB]=(float)rand()/(float)(RAND_MAX)*1.0;
		}
	}
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU


void fillA_poisson(float **A, int n){

	for(int i = 0; i<n*n; i++){
		for(int j = 0; j<n*n; j++){
			if(i==j){
				A[i][j] = 4;
			}else if (i == j+1  || i == j -1 || i==j+n || i==j-n){
				A[i][j] = -1;
			}
		}
	}

	for(int i = n-1; i<n*n-1; i = i + n){
		for(int j = n-1; j<n*n-1; j = j + n){
			A[i+1][j] = 0;
			A[i][j+1] = 0;
		}
	}

}

// struct saxpy_functor
// {
//     const float a;
//
//     saxpy_functor(float _a) : a(_a) {}
//
//     __host__ __device__
//         float operator()(const float& x, const float& y) const {
//             return -1 + (x+y)/(float)((RAND_MAX)*1.0);
//         }
// };
//
// void saxpy_fast(float A, thrust::device_vector<float>& X, thrust::device_vector<float>& Y)
// {
//     // Y <- A * X + Y
//     thrust::transform(X.begin(), X.end(), Y.begin(), Y.begin(), saxpy_functor(A));
// }


void fillB(float *b, int n){
	for(int i =0; i<n; i++)
	{
		b[i] = -1 + 2.0* (float)rand()/(float)((RAND_MAX)*1.0);
	}
}

// void fillB_random_GPU(float *B, int N) {
// 		// Create a pseudo-random number generator
// 		hiprandGenerator_t prng;
// 		hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
//
// 	 // Set the seed for the random number generator using the system clock
// 	 hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());
//
// 	 // Fill the array with random numbers on the device
// 	 hiprandGenerateUniform(prng, B, N);
//
// // 	 float myrandf = hiprand_uniform(&(my_curandstate[idx]));
// // myrandf *= (max_rand_int[idx] - min_rand_int[idx] + 0.999999);
// // myrandf += min_rand_int[idx];
// // int myrand = (int)truncf(myrandf);
//
// }





// Device version of the Jacobi method
__global__ void jacobiOnDevice(float* A, float* b, float* X_New, float* X_Old, int N, float eps){

	unsigned int i, j;
	float sigma = 0, newValue;


	i = threadIdx.x + blockIdx.x * blockDim.x;

	X_Old[i] = X_New[i];

	for (j = 0; j < N; j++) {
		if (i != j) {
			sigma = sigma + A[i*N + j] * X_Old[j];
		}
	}
	//assert(A[i*N+i] != 0);
	newValue = (b[i] - sigma) / A[i*N + i];

	//if (abs(X_Old[i] - newValue) > eps) flag = 0;
	X_New[i] = newValue;
	//newValue;

}


int main(int argc, char* argv[]){
	// int num_devices, device;
	// hipGetDeviceCount(&num_devices);
	// if (num_devices > 1) {
	//   int max_multiprocessors = 0, max_device = 0;
	//   for (device = 0; device < num_devices; device++) {
	//           hipDeviceProp_t properties;
	//           hipGetDeviceProperties(&properties, device);
	//           if (max_multiprocessors < properties.multiProcessorCount) {
	//                   max_multiprocessors = properties.multiProcessorCount;
	//                   max_device = device;
	//           }
	//   }
	//   hipSetDevice(max_device);
	// 	printf("device %d set\n", max_device);
	// }
	hipError_t  cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	timeval t1, t2; // Structs for timing

	float **A, *A_1d, *b;
	float *X_New, *X_Old, *x;

// gpu Copy
float *A_1d_gpu;
float *b_gpu;
float *X_New_gpu;
float *X_Old_gpu;

	srand(0);
	int n = strtol(argv[1], NULL, 10);
	int N = n*n;
	//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-//
	// Set our tolerance and maximum iterations
	float eps = 1.0e-4;
	int maxit = 2*N*N;


	init2d(&A, N);
	fillA_poisson(A, n);

	init1d(&b, N);

	//printf("A Matrix: \n");
	//print(A, N);

	//printf("b Matrix: ");
	//print(b, N);
	init1d(&x, N);



		// STARTING cuda
		//thrust::device_vector<float> b_gpu(N);

		     // Fill the arrays A and B on GPU with random numbers
		  //fillB_random_GPU(thrust::raw_pointer_cast(&b_gpu[0]), N);
			//saxpy_fast(5, b_gpu, b_gpu);

	//fill b
	fillB(b, n);
	//b = thrust::raw_pointer_cast(&b_gpu[0]);
	// thrust::device_ptr<int> dev_ptr = thrust::device_pointer_cast(&b_gpu[0]);
	// for (size_t i = 0; i < N; i++) {
	// 	/* code */
	// 	b[i] = dev_ptr[i];
	// }

	//jacobiSolve(N, A, b, x, eps, maxit);
	//print(x, N);
	//printf("Correct one\n");


		/* ...Convert Matrix_A into 1-D array Input_A ......*/
		A_1d  = (float *)malloc(N*N*sizeof(float));
		//fillA_random_GPU(A,N);
		convertTo1D(A, A_1d, N);

	// on HOST
	//initialize auxiliary data structures
	X_New  = (float *) malloc (N * sizeof(float));
	X_Old  = (float *) malloc (N * sizeof(float));

	// Allocate memory on the device
	 assert(hipSuccess == hipMalloc((void **) &X_New_gpu, N*sizeof(float)));
	 assert(hipSuccess == hipMalloc((void **) &A_1d_gpu, N*N*sizeof(float)));
	 assert(hipSuccess == hipMalloc((void **) &X_Old_gpu, N*sizeof(float)));
	 assert(hipSuccess == hipMalloc((void **) &b_gpu, N*sizeof(float)));

	 hipError_t ct;
	 // Copy data -> device
	 ct = hipMemcpy(X_New_gpu, X_New, sizeof(float)*N, hipMemcpyHostToDevice);
	 assert(ct==hipSuccess);
	 ct = hipMemcpy(A_1d_gpu, A_1d, sizeof(float)*N*N, hipMemcpyHostToDevice);
	 assert(ct==hipSuccess);
	 ct = hipMemcpy(X_Old_gpu, X_Old, sizeof(float)*N, hipMemcpyHostToDevice);
	 assert(ct==hipSuccess);
	 ct = hipMemcpy(b_gpu, b, sizeof(float)*N, hipMemcpyHostToDevice);
	 assert(ct==hipSuccess);


	gettimeofday(&t1, NULL);

	int block_size = strtol(argv[2], NULL, 10);
  int n_blocks = N/block_size + (N%block_size == 0 ? 0:1);
  //square_array <<< n_blocks, block_size >>> (a_d, N);

	int Iteration = 0;
  hipDeviceSynchronize();
	//nt cpuConvergenceTest = 0;
	do{
		// cpuConvergenceTest = 1;
		// hipMemcpyToSymbol(HIP_SYMBOL(flag), &cpuConvergenceTest, sizeof(int));

		//#error Add GPU kernel calls here (see CPU version above)
		//jacobiOnDevice <<< 1, N >>> (A_1d_gpu, thrust::raw_pointer_cast(&b_gpu[0]), X_New_gpu, X_Old_gpu, N, eps);
		jacobiOnDevice <<< n_blocks, block_size >>> (A_1d_gpu,b_gpu, X_New_gpu, X_Old_gpu, N, eps);

		//jacobi<<16,1>>

		hipError_t cudaStatus = hipDeviceSynchronize();
				if (cudaStatus != hipSuccess) {
					printf("hipDeviceSynchronize returned error code %d after launching jacobi!\n", cudaStatus);
				}

				//hipMemcpyFromSymbol(&cpuConvergenceTest, HIP_SYMBOL(flag), sizeof(int));

		Iteration += 1;
		//hipDeviceSynchronize();
		hipMemcpy(X_New, X_New_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
		hipMemcpy(X_Old, X_Old_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);

	}while( (Iteration < maxit) && getError(X_Old, X_New, N) >= eps);
	//}while( (Iteration < maxit) &&  getErrorThrust<float>(X_Old_gpu, X_New_gpu, N) >= eps);
	//hipMemcpy(X_Old, X_Old_gpu, sizeof(float)*N, hipMemcpyDeviceToHost);
	//print(X_New, N);
	// Data <- device

    // Free memory
  hipFree(X_New_gpu);
  hipFree(X_Old_gpu);
	hipFree(b_gpu);
	hipFree(A_1d_gpu);
	free(X_Old);
  free(X_New);
  free(A_1d);
  free(A);
  free(b);


	hipDeviceSynchronize();

	gettimeofday(&t2, NULL);


	printf("%g \n",
	               t2.tv_sec - t1.tv_sec +
	               (t2.tv_usec - t1.tv_usec) / 1.0e6);

	return 0;
}
